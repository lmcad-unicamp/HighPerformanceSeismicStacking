#include "common/include/output/Logger.hpp"
#include "cuda/include/execution/CudaUtils.hpp"

#include <hip/hip_runtime.h>
#include <sstream>
#include <stdexcept>

void cudaAssert(hipError_t errorCode, const char *file, int line) {
    if (errorCode != hipSuccess) {
        ostringstream stringStream;
        stringStream << "CUDA error detected at " << file << "::" << line << " with message " << hipGetErrorString(errorCode);
        LOGE(stringStream.str());
        throw runtime_error(stringStream.str());
    }
}
