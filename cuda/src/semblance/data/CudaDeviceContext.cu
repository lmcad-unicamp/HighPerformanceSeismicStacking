#include "cuda/include/semblance/data/CudaDeviceContext.hpp"

#include <hip/hip_runtime.h>
#include <sstream>
#include <stdexcept>

CudaDeviceContext::CudaDeviceContext(unsigned int devId) : DeviceContext(devId) {
}

void CudaDeviceContext::activate() const {
    hipError_t errorCode = hipSetDevice(devId);

    if (errorCode != hipSuccess) {
        ostringstream stringStream;
        stringStream << "Creating CUDA hipSetDevice failed with error " << errorCode;
        throw runtime_error(stringStream.str());
    }
}