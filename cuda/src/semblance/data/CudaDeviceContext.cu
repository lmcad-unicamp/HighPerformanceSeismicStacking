#include "cuda/include/semblance/data/CudaDeviceContext.hpp"

#include <hip/hip_runtime.h>
#include <sstream>
#include <stdexcept>

using namespace std;

CudaDeviceContext::CudaDeviceContext(unsigned int devId) : DeviceContext(devId) {
}

void CudaDeviceContext::activate() const {
    hipError_t errorCode = hipSetDevice(deviceId);

    if (errorCode != hipSuccess) {
        ostringstream stringStream;
        stringStream << "Creating CUDA hipSetDevice failed with error " << errorCode;
        throw runtime_error(stringStream.str());
    }
}
