#include "hip/hip_runtime.h"
#include "common/include/gpu/interface.h"
#include "cuda/include/semblance/kernel/zocrs/linear_search.cuh"

__global__
void buildParameterArrayForZeroOffsetCommonReflectionSurface(
    float* parameterArray,
    float minVelocity,
    float incrementVelocity,
    unsigned int countVelocity,
    float minA,
    float incrementA,
    unsigned int countA,
    float minB,
    float incrementB,
    unsigned int countB,
    unsigned int totalParameterCount
) {
    unsigned int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadIndex < totalParameterCount) {
        unsigned int idxVelocity = (threadIndex / (countA * countB)) % countVelocity;
        unsigned int idxA = (threadIndex / countB) % countA;
        unsigned int idxB = threadIndex % countB;

        float v = minVelocity + static_cast<float>(idxVelocity) * incrementVelocity;
        float a = minA + static_cast<float>(idxA) * incrementA;
        float b = minB + static_cast<float>(idxB) * incrementB;

        unsigned int offset = 3 * threadIndex;
        parameterArray[offset] = 4.0f / (v * v);
        parameterArray[offset + 1] = a;
        parameterArray[offset + 2] = b;
    }
}

__global__
void computeSemblancesForZeroOffsetCommonReflectionSurface(
    const float *samples,
    const float *midpoint,
    const float *halfoffsetSquared,
    unsigned int traceCount,
    unsigned int samplesPerTrace,
    float m0,
    float dtInSeconds,
    int tauIndexDisplacement,
    int windowSize,
    /* Parameter arrays */
    const float *parameterArray,
    unsigned int totalParameterCount,
    /* Output arrays */
    float *semblanceArray,
    float *stackArray
) {
    unsigned int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    unsigned int sampleIndex = threadIndex / totalParameterCount;
    unsigned int parameterIndex = threadIndex % totalParameterCount;
    unsigned int parameterOffset = 3 * parameterIndex;

    if (sampleIndex < samplesPerTrace) {
        float semblance = 0;
        float stack = 0;

        float t0 = static_cast<float>(sampleIndex) * dtInSeconds;

        float c = parameterArray[parameterOffset];
        float a = parameterArray[parameterOffset + 1];
        float b = parameterArray[parameterOffset + 2];

        float numeratorComponents[MAX_WINDOW_SIZE];
        float denominatorSum = 0;
        float linearSum = 0;

        for (unsigned int i = 0; i < MAX_WINDOW_SIZE; i++) {
            numeratorComponents[i] = 0;
        }

        unsigned int usedCount = 0;

        for (unsigned int traceIndex = 0; traceIndex < traceCount; traceIndex++) {
            float m = midpoint[traceIndex];
            float h_sq = halfoffsetSquared[traceIndex];
            const float *traceSamples = samples + traceIndex * samplesPerTrace;

            float dm = m - m0;
            float tmp = t0 + a * dm;
            tmp = tmp * tmp + b * dm * dm + c * h_sq;

            if (tmp >= 0) {
                float tIndex = sqrt(tmp)/ dtInSeconds;
                int kIndex = static_cast<int>(tIndex);
                float dt = tIndex - static_cast<float>(kIndex);
            
                if ((kIndex - tauIndexDisplacement >= 0) &&
                    (kIndex + tauIndexDisplacement + 1 < static_cast<int>(samplesPerTrace))) {
                    
                    int k = kIndex - tauIndexDisplacement;
                    float u, y0, y1;
                    
                    y1 = traceSamples[k];
                    
                    for (int j = 0; j < windowSize; j++, k++) {
                        y0 = y1;
                        y1 = traceSamples[k + 1];
                        u = (y1 - y0) * dt + y0;
            
                        numeratorComponents[j] += u;
                        linearSum += u;
                        denominatorSum += u * u;
                    }
            
                    usedCount++;
                }
            }
        }

        if (usedCount > 0) {
            float sumNumerator = 0;
            for (int w = 0; w < windowSize; w++) {
                sumNumerator += numeratorComponents[w] * numeratorComponents[w];
            }

            semblance = sumNumerator / (usedCount * denominatorSum);
            stack = linearSum / (usedCount * windowSize);
        }

        unsigned int offset = sampleIndex * totalParameterCount + parameterIndex;
        semblanceArray[offset] = semblance;
        stackArray[offset] = stack;
    }
}

__global__
void selectBestSemblancesForZeroOffsetCommonReflectionSurface(
    const float *semblanceArray,
    const float *stackArray,
    const float *parameterArray,
    unsigned int totalParameterCount,
    unsigned int samplesPerTrace,
    float *resultArray
) {
    unsigned int sampleIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (sampleIndex < samplesPerTrace) {
        unsigned int offset = sampleIndex * totalParameterCount;

        float bestSemblance = -1, bestStack, bestVelocity, bestA, bestB;

        for (unsigned int parameterIndex = 0; parameterIndex < totalParameterCount; parameterIndex++) {
            unsigned int offsetParameter = parameterIndex * 3;

            float semblance = semblanceArray[offset + parameterIndex];
            float stack = stackArray[offset + parameterIndex];
            float c = parameterArray[offsetParameter];
            float a = parameterArray[offsetParameter + 1];
            float b = parameterArray[offsetParameter + 2];

            if (semblance > bestSemblance) {
                bestSemblance = semblance;
                bestStack = stack;
                bestVelocity = 2 / sqrt(c);
                bestA = a;
                bestB = b;
            }
        }

        resultArray[sampleIndex] = bestSemblance;
        resultArray[samplesPerTrace + sampleIndex] = bestStack;
        resultArray[2 * samplesPerTrace + sampleIndex] = bestVelocity;
        resultArray[3 * samplesPerTrace + sampleIndex] = bestA;
        resultArray[4 * samplesPerTrace + sampleIndex] = bestB;
    }
}

__global__
void selectTracesForZeroOffsetCommonReflectionSurface(
    const float* midpointArray,
    unsigned int traceCount,
    unsigned char* usedTraceMaskArray,
    float m0,
    float apm
) {
    unsigned int traceIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (traceIndex < traceCount) {
        usedTraceMaskArray[traceIndex] = fabs(m0 - midpointArray[traceIndex]) <= apm;
    }
}
