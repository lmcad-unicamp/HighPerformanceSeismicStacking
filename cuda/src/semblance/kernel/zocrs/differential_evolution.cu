#include "hip/hip_runtime.h"
#include "common/include/gpu/interface.h"
#include "cuda/include/semblance/kernel/zocrs/differential_evolution.cuh"

__global__
void computeSemblancesForZeroOffsetCommonReflectionSurface(
    const float *samples,
    const float *midpoint,
    const float *halfoffsetSquared,
    unsigned int traceCount,
    unsigned int samplesPerTrace,
    unsigned int individualsPerPopulation,
    float m0,
    float dtInSeconds,
    int tauIndexDisplacement,
    int windowSize,
    unsigned int numberOfCommonResults,
    const float *x,
    float *fx
) {
    unsigned int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int sampleIndex = threadIndex / individualsPerPopulation;
    unsigned int individualIndex = threadIndex % individualsPerPopulation;

    if (sampleIndex < samplesPerTrace) {
        float semblance = 0;
        float stack = 0;

        unsigned int parameterOffset = (sampleIndex * individualsPerPopulation + individualIndex) * 3;

        float t0 = static_cast<float>(sampleIndex) * dtInSeconds;

        float v = x[parameterOffset];
        float c = 4.0f / (v * v);
        float a = x[parameterOffset + 1];
        float b = x[parameterOffset + 2];

        float numeratorComponents[MAX_WINDOW_SIZE];
        float denominatorSum = 0;
        float linearSum = 0;

        for (unsigned int i = 0; i < MAX_WINDOW_SIZE; i++) {
            numeratorComponents[i] = 0;
        }

        unsigned int usedCount = 0;

        for (unsigned int traceIndex = 0; traceIndex < traceCount; traceIndex++) {
            float m = midpoint[traceIndex];
            float h_sq = halfoffsetSquared[traceIndex];
            const float *traceSamples = samples + traceIndex * samplesPerTrace;

            float dm = m - m0;
            float tmp = t0 + a * dm;
            tmp = tmp * tmp + b * dm * dm + c * h_sq;

            if (tmp >= 0) {
                float tIndex = sqrt(tmp)/ dtInSeconds;
                int kIndex = static_cast<int>(tIndex);
                float dt = tIndex - static_cast<float>(kIndex);
            
                if ((kIndex - tauIndexDisplacement >= 0) &&
                    (kIndex + tauIndexDisplacement + 1 < static_cast<int>(samplesPerTrace))) {
                    
                    int k = kIndex - tauIndexDisplacement;
                    float u, y0, y1;
                    
                    y1 = traceSamples[k];
                    
                    for (int j = 0; j < windowSize; j++, k++) {
                        y0 = y1;
                        y1 = traceSamples[k + 1];
                        u = (y1 - y0) * dt + y0;
            
                        numeratorComponents[j] += u;
                        linearSum += u;
                        denominatorSum += u * u;
                    }
            
                    usedCount++;
                }
            }
        }

        if (usedCount > 0) {
            float sumNumerator = 0;
            for (int w = 0; w < windowSize; w++) {
                sumNumerator += numeratorComponents[w] * numeratorComponents[w];
            }

            semblance = sumNumerator / (usedCount * denominatorSum);
            stack = linearSum / (usedCount * windowSize);
        }

        unsigned int offset = sampleIndex * individualsPerPopulation * numberOfCommonResults;
        fx[offset] = semblance;
        fx[offset + 1] = stack;
    }
}

__global__
void selectBestIndividualsForZeroOffsetCommonReflectionSurface(
    const float* x,
    const float* fx,
    float* resultArray,
    unsigned int individualsPerPopulation,
    unsigned int samplesPerTrace,
    unsigned int numberOfCommonResults
) {
    unsigned int sampleIndex = threadIdx.x + blockIdx.x * blockDim.x;

    if (sampleIndex < samplesPerTrace) {
        unsigned int popIndex = sampleIndex * individualsPerPopulation * 3;
        unsigned int fitnessIndex = sampleIndex * individualsPerPopulation * numberOfCommonResults;

        float bestSemblance = -1, bestStack, bestVelocity, bestA, bestB;

        for (unsigned int individualIndex = 0; individualIndex < individualsPerPopulation; individualIndex++) {
            unsigned int featureOffset = fitnessIndex + individualIndex * numberOfCommonResults;
            unsigned int individualOffset = popIndex + 3 * individualIndex;

            float semblance = fx[featureOffset];
            float stack = fx[featureOffset + 1];
            float velocity = x[individualOffset];
            float a = x[individualOffset + 1];
            float b = x[individualOffset + 2];

            if (semblance > bestSemblance) {
                bestSemblance = semblance;
                bestStack = stack;
                bestVelocity = velocity;
                bestA = a;
                bestB = b;
            }
        }

        resultArray[sampleIndex] = bestSemblance;
        resultArray[samplesPerTrace + sampleIndex] = bestStack;
        resultArray[2 * samplesPerTrace + sampleIndex] = bestVelocity;
        resultArray[3 * samplesPerTrace + sampleIndex] = bestA;
        resultArray[4 * samplesPerTrace + sampleIndex] = bestB;
    }
}
