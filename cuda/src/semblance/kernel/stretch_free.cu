#include "hip/hip_runtime.h"
#include "cuda/include/semblance/kernel/base.h"
#include "cuda/include/semblance/kernel/stretch_free.h"

__global__
void kernelStretchFree( const float *samples,
                        const float *midpoint,
                        const float *halfoffset,
                        unsigned int traceCount,
                        const float *inputParameters,
                        gpu_gather_data_t gatherData,
                        gpu_reference_point_t referencePoint,
                        gpu_traveltime_data_t traveltime,
                        /* Parameter arrays */
                        const float *parameterArray,
                        /* Output arrays */
                        float *resultArray,
                        /* Missed traces array */
                        float* notUsedCountArray
) {
    extern __shared__ float threadSemblanceData[];

    unsigned int arrayStep = blockDim.x;

    unsigned int parameterThreadIndex = threadIdx.x;
    unsigned int sampleIndex = blockIdx.x;
    unsigned int notUsedIndex = blockIdx.x * blockDim.x + threadIdx.x;

    unsigned int numberOfParameters = traveltime.numberOfParameters;
    unsigned int numberOfCommonResults = traveltime.numberOfCommonResults;
    unsigned int numberOfTotalParameters =
        numberOfParameters + numberOfCommonResults;

    unsigned int samplesPerTrace = gatherData.samplesPerTrace;

    if (sampleIndex < samplesPerTrace) {

        gpu_semblance_compute_data_t semblanceCompute;
        gpu_traveltime_parameter_t travelTimeThreadData, stretchData;
        gpu_reference_point_t stretchReferencePoint = referencePoint;

        unsigned int notUsedCount = 0;
        unsigned int usedCount = 0;

        int n = static_cast<int>(parameterArray[parameterThreadIndex]);

        threadSemblanceData[parameterThreadIndex] = 0;
        threadSemblanceData[arrayStep + parameterThreadIndex] = 0;
        threadSemblanceData[2 * arrayStep + parameterThreadIndex] = parameterArray[parameterThreadIndex];

        if ((static_cast<int>(sampleIndex) - n) >= 0 && (static_cast<int>(sampleIndex) - n) < samplesPerTrace) {

            referencePoint.t0 = sampleIndex * gatherData.dtInSeconds;
            stretchReferencePoint.t0 = (sampleIndex - n) * gatherData.dtInSeconds;

            travelTimeThreadData.numberOfParameters = numberOfParameters;
            stretchData.numberOfParameters = numberOfParameters;

            for (unsigned int parameterIndex = 0;
                parameterIndex < numberOfParameters;
                parameterIndex++) {

                unsigned int step = parameterIndex * samplesPerTrace;
                travelTimeThreadData.semblanceParameters[parameterIndex] = inputParameters[step + sampleIndex];
                stretchData.semblanceParameters[parameterIndex] = inputParameters[step + sampleIndex - n];
            }

            semblanceCompute.denominatorSum = semblanceCompute.linearSum = 0;
            memset(semblanceCompute.numeratorComponents, 0, MAX_WINDOW_SIZE * sizeof(float));

            for (unsigned int traceIndex = 0; traceIndex < traceCount; traceIndex++) {

                float t;
                float h = halfoffset[traceIndex];
                float m = midpoint[traceIndex];
                const float *traceSamples = samples + traceIndex * gatherData.samplesPerTrace;

                if (traveltime.traveltime == OCT) {
                    if (!shouldUseTrace(
                            m, h,
                            gatherData,
                            referencePoint,
                            traveltime,
                            travelTimeThreadData,
                            &travelTimeThreadData.mh)
                        ) {
                        notUsedCount++;
                        continue;
                    }
                }

                if (computeTime(m, h, stretchReferencePoint, traveltime, stretchData, &t) == NO_ERROR) {

                    float tStretch = t + static_cast<float>(n) * gatherData.dtInSeconds;

                    if (computeSemblance(traceSamples, tStretch, gatherData, &semblanceCompute) == NO_ERROR) {
                        usedCount++;
                    }
                }
            }

            if (usedCount) {

                float sumNumerator = 0;
                for (int j = 0; j < gatherData.windowSize; j++) {
                    sumNumerator += semblanceCompute.numeratorComponents[j] * semblanceCompute.numeratorComponents[j];
                }

                threadSemblanceData[parameterThreadIndex] =
                    sumNumerator / (usedCount * semblanceCompute.denominatorSum);
                threadSemblanceData[arrayStep + parameterThreadIndex] =
                    semblanceCompute.linearSum / (usedCount * gatherData.windowSize);
            }

            notUsedCountArray[notUsedIndex] += notUsedCount;
        }

        __syncthreads();

        /* Reduce the best results */
        for (unsigned int s = blockDim.x / 2; s > 0; s = s >> 1) {
            if (parameterThreadIndex < s) {
                if (threadSemblanceData[parameterThreadIndex] < threadSemblanceData[parameterThreadIndex + s]) {
                    for (unsigned int i = 0; i < numberOfTotalParameters; i++) {
                        unsigned step = i * arrayStep;
                        threadSemblanceData[step + parameterThreadIndex] =
                            threadSemblanceData[step + parameterThreadIndex + s];
                    }
                }
            }
            __syncthreads();
        }

        if (parameterThreadIndex == 0) {
            if (threadSemblanceData[0] > resultArray[sampleIndex]) {
                for (unsigned int i = 0; i < numberOfTotalParameters; i++) {
                    unsigned int step = i * samplesPerTrace;
                    resultArray[step + sampleIndex] = threadSemblanceData[step];
                }
            }
        }
    }
}

__global__
void filterOutTracesForOffsetContinuationTrajectoryAndStretchFree(
    const float *midpointArray,
    const float *halfoffsetArray,
    unsigned char* mustUseTraceArray,
    unsigned int traceCount,
    gpu_gather_data_t gatherData,
    gpu_reference_point_t referencePoint,
    gpu_traveltime_data_t traveltime,
    const float *parameterArray,
    unsigned int parameterCount
) {
    unsigned int traceIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (traceIndex < traceCount) {

        gpu_traveltime_parameter_t travelTimeThreadData;

        float m, h;
        m = midpointArray[traceIndex];
        h = halfoffsetArray[traceIndex];

        mustUseTraceArray[traceIndex] = 0;

        for (unsigned int sampleIndex = 0; !mustUseTraceArray[traceIndex] && sampleIndex < gatherData.samplesPerTrace; sampleIndex++) {

            referencePoint.t0 = sampleIndex * gatherData.dtInSeconds;

            for (unsigned int parameterIndex = 0; !mustUseTraceArray[traceIndex] && parameterIndex < parameterCount; parameterIndex++) {

                for (unsigned int i = 0; i < traveltime.numberOfParameters; i++) {
                    unsigned int step = i * parameterCount;
                    travelTimeThreadData.semblanceParameters[i] = parameterArray[step + parameterIndex];
                }

                mustUseTraceArray[traceIndex] =
                    shouldUseTrace(
                        m,
                        h,
                        gatherData,
                        referencePoint,
                        traveltime,
                        travelTimeThreadData,
                        &travelTimeThreadData.mh
                    );
            }
        }
    }
}
