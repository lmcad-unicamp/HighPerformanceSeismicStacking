#include "hip/hip_runtime.h"
#include "common/include/gpu/interface.h"
#include "cuda/include/semblance/kernel/cmp/differential_evolution.cuh"

__global__
void computeSemblancesForCommonMidPoint(
    const float *samples,
    const float *halfoffsetSquared,
    unsigned int startingTraceIndex,
    unsigned int traceCount,
    unsigned int samplesPerTrace,
    unsigned int individualsPerPopulation,
    float dtInSeconds,
    int tauIndexDisplacement,
    int windowSize,
    unsigned int numberOfCommonResults,
    const float *x,
    float *fx
) {
    unsigned int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int sampleIndex = threadIndex / individualsPerPopulation;
    unsigned int individualIndex = threadIndex % individualsPerPopulation;

    if (sampleIndex < samplesPerTrace) {
        float semblance = 0;
        float stack = 0;

        unsigned int parameterIndex = sampleIndex * individualsPerPopulation + individualIndex;

        float t0 = static_cast<float>(sampleIndex) * dtInSeconds;
        float v = x[parameterIndex];
        float c = 4.0f / (v * v);

        float numeratorComponents[MAX_WINDOW_SIZE];
        float denominatorSum = 0;
        float linearSum = 0;

        RESET_SEMBLANCE_NUM_COMP(numeratorComponents, MAX_WINDOW_SIZE);

        unsigned int usedCount = 0;

        for (unsigned int traceIndex = 0; traceIndex < traceCount; traceIndex++) {
            unsigned int traceIndexWithOffset = startingTraceIndex + traceIndex;

            float h_sq = halfoffsetSquared[traceIndexWithOffset];
            const float *traceSamples = samples + traceIndexWithOffset * samplesPerTrace;

            float t = sqrt(t0 * t0 + c * h_sq);

            COMPUTE_SEMBLANCE(
                t,
                dtInSeconds,
                samplesPerTrace,
                tauIndexDisplacement,
                windowSize,
                numeratorComponents,
                linearSum,
                denominatorSum,
                usedCount
            );
        }

        REDUCE_SEMBLANCE_STACK(numeratorComponents, linearSum, denominatorSum, windowSize, usedCount, semblance, stack);

        unsigned int offset = (sampleIndex * individualsPerPopulation + individualIndex) * numberOfCommonResults;
        fx[offset] = semblance;
        fx[offset + 1] = stack;
    }
}

__global__
void selectBestIndividualsForCommonMidPoint(
    const float* x,
    const float* fx,
    float* resultArray,
    unsigned int individualsPerPopulation,
    unsigned int samplesPerTrace,
    unsigned int numberOfCommonResults
) {
    unsigned int sampleIndex = threadIdx.x + blockIdx.x * blockDim.x;

    if (sampleIndex < samplesPerTrace) {
        unsigned int popIndex = sampleIndex * individualsPerPopulation;
        unsigned int fitnessIndex = sampleIndex * individualsPerPopulation * numberOfCommonResults;

        float bestSemblance = -1, bestStack, bestVelocity;

        for (unsigned int individualIndex = 0; individualIndex < individualsPerPopulation; individualIndex++) {
            unsigned int featureOffset = fitnessIndex + individualIndex * numberOfCommonResults;

            float semblance = fx[featureOffset];
            float stack = fx[featureOffset + 1];
            float velocity = x[popIndex + individualIndex];

            if (semblance > bestSemblance) {
                bestSemblance = semblance;
                bestStack = stack;
                bestVelocity = velocity;
            }
        }

        resultArray[sampleIndex] = bestSemblance;
        resultArray[samplesPerTrace + sampleIndex] = bestStack;
        resultArray[2 * samplesPerTrace + sampleIndex] = bestVelocity;
    }
}
