#include "hip/hip_runtime.h"
#include "common/include/gpu/interface.h"
#include "cuda/include/semblance/kernel/cmp/stretch_free.cuh"

__global__
void computeSemblancesForCommonMidPoint(
    const float *samples,
    const float *halfoffsetSquared,
    unsigned int traceCount,
    unsigned int samplesPerTrace,
    float dtInSeconds,
    int tauIndexDisplacement,
    int windowSize,
    /* Parameter arrays */
    const float *parameterArray,
    const float *nArray,
    unsigned int totalNCount,
    /* Output arrays */
    float *semblanceArray,
    float *stackArray
) {
    unsigned int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    unsigned int sampleIndex = threadIndex / totalNCount;
    unsigned int nIndex = threadIndex % totalNCount;

    if (sampleIndex < samplesPerTrace) {
        float semblance = 0;
        float stack = 0;

        int n = static_cast<int>(nArray[nIndex]);
        int sampleIndex_n = static_cast<int>(sampleIndex) - n;

        if (sampleIndex_n >= 0 && sampleIndex_n < samplesPerTrace) {

            float t0_n = static_cast<float>(sampleIndex_n) * dtInSeconds;

            float v_n = parameterArray[sampleIndex_n];
            float c_n = 4.0f / (v_n * v_n);

            float numeratorComponents[MAX_WINDOW_SIZE];
            float denominatorSum = 0;
            float linearSum = 0;

            RESET_SEMBLANCE_NUM_COMP(numeratorComponents, MAX_WINDOW_SIZE);

            unsigned int usedCount = 0;

            for (unsigned int traceIndex = 0; traceIndex < traceCount; traceIndex++) {
                float h_sq = halfoffsetSquared[traceIndex];
                const float *traceSamples = samples + traceIndex * samplesPerTrace;

                float t_n = sqrt(t0_n * t0_n + c_n * h_sq);

                float t = t_n + static_cast<float>(n) * dtInSeconds;

                COMPUTE_SEMBLANCE(t, dtInSeconds, samplesPerTrace, tauIndexDisplacement, windowSize, numeratorComponents, linearSum, denominatorSum, usedCount);
            }

            REDUCE_SEMBLANCE_STACK(numeratorComponents, linearSum, denominatorSum, windowSize, usedCount, semblance, stack);
        }

        unsigned int offset = sampleIndex * totalNCount + nIndex;
        semblanceArray[offset] = semblance;
        stackArray[offset] = stack;
    }
}