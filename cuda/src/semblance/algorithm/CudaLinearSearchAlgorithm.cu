#include "cuda/include/semblance/algorithm/CudaLinearSearchAlgorithm.hpp"
#include "cuda/include/semblance/data/container.hpp"
#include "cuda/include/semblance/kernel/base.h"
#include "cuda/include/semblance/kernel/linear_search.h"

#include <numeric>
#include <sstream>
#include <stdexcept>

using namespace std;

CudaLinearSearchAlgorithm::CudaLinearSearchAlgorithm(
    shared_ptr<Traveltime> traveltime,
    shared_ptr<DeviceContext> context,
    DataContainerBuilder* dataBuilder
) : LinearSearchAlgorithm(model, context, dataBuilder) {
}

void CudaLinearSearchAlgorithm::computeSemblanceAtGpuForMidpoint(float m0) {
    Gather* gather = Gather::getInstance();

    dim3 dimGrid(gather->getSamplesPerTrace());

    gpu_gather_data_t kernelData = gather->getGpuGatherData();

    gpu_traveltime_data_t kerneltraveltime = traveltime->toGpuData();

    gpu_reference_point_t kernelReferencePoint;
    kernelReferencePoint.m0 = m0;
    kernelReferencePoint.h0 = traveltime->getReferenceHalfoffset();

    unsigned int sharedMemSizeCount =
        traveltime->getNumberOfResults() * threadCount * static_cast<unsigned int>(sizeof(float));

    kernelLinearSearch<<< dimGrid, threadCount, sharedMemSizeCount >>>(
        CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::FILT_SAMPL]),
        CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::FILT_MDPNT]),
        CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::FILT_HLFOFFST]),
        filteredTracesCount,
        kernelData,
        kernelReferencePoint,
        kerneltraveltime,
        CUDA_DEV_PTR(deviceParameterArray),
        CUDA_DEV_PTR(deviceResultArray),
        CUDA_DEV_PTR(deviceNotUsedCountArray)
    );

    hipError_t errorCode = hipGetLastError();

    if (errorCode != hipSuccess) {
        ostringstream stringStream;
        stringStream << "Creating CUDA kernelLinearSearch<<<>>> launch failed with error " << errorCode;
        throw runtime_error(stringStream.str());
    }

    hipDeviceSynchronize();
}

void CudaLinearSearchAlgorithm::selectTracesToBeUsedForMidpoint(float m0) {

    Gather* gather = Gather::getInstance();

    unsigned int traceCount = gather->getTotalTracesCount();

    vector<unsigned char> usedTraceMask(traceCount);

    unsigned char* deviceUsedTraceMaskArray;
    hipMalloc((void **) &deviceUsedTraceMaskArray, traceCount * sizeof(char));

    dim3 dimGrid(traceCount / threadCount + 1);

    gpu_gather_data_t gatherData = gather->getGpuGatherData();

    gpu_traveltime_data_t kerneltraveltime = traveltime->toGpuData();

    gpu_reference_point_t kernelReferencePoint;
    kernelReferencePoint.m0 = m0;
    kernelReferencePoint.h0 = traveltime->getReferenceHalfoffset();

    switch (traveltime->getModel()) {
        case CMP:
        case ZOCRS:
            filterMidpointDependentTraces<<<dimGrid, threadCount>>>(
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::MDPNT]),
                traceCount,
                deviceUsedTraceMaskArray,
                traveltime->toGpuData(),
                gather->getApm(),
                m0
            );
            break;

        case OCT:
            filterOutTracesForOffsetContinuationTrajectoryAndLinearSearch<<<dimGrid, threadCount>>>(
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::MDPNT]),
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::HLFOFFST]),
                deviceUsedTraceMaskArray,
                traceCount,
                gatherData,
                kernelReferencePoint,
                kerneltraveltime,
                CUDA_DEV_PTR(deviceParameterArray),
                getParameterArrayStep()
            );
            break;
    }

    hipError_t errorCode = hipGetLastError();

    if (errorCode != hipSuccess) {
        ostringstream stringStream;
        stringStream << "Creating CUDA kernel launch failed with error " << errorCode;
        throw runtime_error(stringStream.str());
    }

    hipDeviceSynchronize();

    hipMemcpy(usedTraceMask.data(), deviceUsedTraceMaskArray, traceCount * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipFree(deviceUsedTraceMaskArray);

    copyOnlySelectedTracesToDevice(usedTraceMask);
}
