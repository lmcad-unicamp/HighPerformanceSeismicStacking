#include "hip/hip_runtime.h"
#include "common/include/execution/Utils.hpp"
#include "common/include/output/Logger.hpp"
#include "cuda/include/execution/CudaUtils.hpp"
#include "cuda/include/semblance/algorithm/CudaDifferentialEvolutionAlgorithm.hpp"
#include "cuda/include/semblance/data/CudaDataContainer.hpp"
#include "cuda/include/semblance/kernel/common/differential_evolution.cuh"
#include "cuda/include/semblance/kernel/cmp/common.cuh"
#include "cuda/include/semblance/kernel/cmp/differential_evolution.cuh"
#include "cuda/include/semblance/kernel/zocrs/common.cuh"
#include "cuda/include/semblance/kernel/zocrs/differential_evolution.cuh"
#include "cuda/include/semblance/kernel/oct/differential_evolution.cuh"

#include <cmath>
#include <sstream>
#include <stdexcept>
#include <stdlib.h>
#include <time.h>

using namespace std;

CudaDifferentialEvolutionAlgorithm::CudaDifferentialEvolutionAlgorithm(
    shared_ptr<Traveltime> model,
    shared_ptr<DeviceContext> context,
    DataContainerBuilder* dataBuilder,
    unsigned int gen,
    unsigned int ind
) : DifferentialEvolutionAlgorithm(model, context, dataBuilder, gen, ind) {
}

CudaDifferentialEvolutionAlgorithm::~CudaDifferentialEvolutionAlgorithm() {
    CUDA_ASSERT(hipFree(st));
}

void CudaDifferentialEvolutionAlgorithm::computeSemblanceAtGpuForMidpoint(float m0) {

    unsigned int numberOfCommonResults = traveltime->getNumberOfCommonResults();

    Gather* gather = Gather::getInstance();

    unsigned int samplesPerTrace = gather->getSamplesPerTrace();
    float dtInSeconds = gather->getSamplePeriodInSeconds();
    int tauIndexDisplacement = gather->getTauIndexDisplacement();
    unsigned int windowSize = gather->getWindowSize();
    float apm = gather->getApm();    
    float h0 = traveltime->getReferenceHalfoffset();

    dim3 dimGrid(static_cast<int>(ceil(static_cast<float>(individualsPerPopulation * samplesPerTrace) / static_cast<float>(threadCount))));

    switch (traveltime->getModel()) {
        case CMP:
            computeSemblancesForCommonMidPoint<<< dimGrid, threadCount >>>(
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::FILT_SAMPL]),
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::FILT_HLFOFFST_SQ]),
                filteredTracesCount,
                samplesPerTrace,
                individualsPerPopulation,
                dtInSeconds,
                tauIndexDisplacement,
                windowSize,
                numberOfCommonResults,
                CUDA_DEV_PTR(deviceParameterArray),
                CUDA_DEV_PTR(deviceResultArray)
            );
            break;
        case ZOCRS:
            computeSemblancesForZeroOffsetCommonReflectionSurface<<< dimGrid, threadCount >>>(
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::FILT_SAMPL]),
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::FILT_MDPNT]),
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::FILT_HLFOFFST_SQ]),
                filteredTracesCount,
                samplesPerTrace,
                individualsPerPopulation,
                m0,
                dtInSeconds,
                tauIndexDisplacement,
                windowSize,
                numberOfCommonResults,
                CUDA_DEV_PTR(deviceParameterArray),
                CUDA_DEV_PTR(deviceResultArray)
            );
            break;
        case OCT:
            computeSemblancesForOffsetContinuationTrajectory<<< dimGrid, threadCount >>>(
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::FILT_SAMPL]),
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::FILT_MDPNT]),
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::FILT_HLFOFFST]),
                filteredTracesCount,
                samplesPerTrace,
                individualsPerPopulation,
                apm,
                m0,
                h0,
                dtInSeconds,
                tauIndexDisplacement,
                windowSize,
                numberOfCommonResults,
                CUDA_DEV_PTR(deviceNotUsedCountArray),
                CUDA_DEV_PTR(deviceParameterArray),
                CUDA_DEV_PTR(deviceResultArray)
            );
            break;
        default:
            throw invalid_argument("Invalid traveltime model");
    }

    CUDA_ASSERT(hipDeviceSynchronize());

    CUDA_ASSERT(hipGetLastError());
}

void CudaDifferentialEvolutionAlgorithm::selectTracesToBeUsedForMidpoint(float m0) {

    LOGI("Selecting traces for m0 = " << m0);

    Gather* gather = Gather::getInstance();

    unsigned int traceCount = gather->getTotalTracesCount();

    unsigned int samplesPerTrace = gather->getSamplesPerTrace();
    float dtInSeconds = gather->getSamplePeriodInSeconds();
    int tauIndexDisplacement = gather->getTauIndexDisplacement();
    unsigned int windowSize = gather->getWindowSize();
    float apm = gather->getApm();    
    float h0 = traveltime->getReferenceHalfoffset();

    vector<unsigned char> usedTraceMask(traceCount);

    unsigned char* deviceUsedTraceMaskArray;
    CUDA_ASSERT(hipMalloc((void **) &deviceUsedTraceMaskArray, traceCount * sizeof(char)));
    CUDA_ASSERT(hipMemset(deviceUsedTraceMaskArray, 0, traceCount * sizeof(unsigned char)))

    dim3 dimGrid(static_cast<int>(ceil(static_cast<float>(traceCount) / static_cast<float>(threadCount))));

    chrono::duration<double> copyTime = chrono::duration<double>::zero();

    switch (traveltime->getModel()) {
        case CMP:
            selectTracesForCommonMidPoint<<<dimGrid, threadCount>>>(
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::MDPNT]),
                traceCount,
                deviceUsedTraceMaskArray,
                m0
            );
            break;
        case ZOCRS:
            selectTracesForZeroOffsetCommonReflectionSurface<<<dimGrid, threadCount>>>(
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::MDPNT]),
                traceCount,
                deviceUsedTraceMaskArray,
                m0,
                apm
            );
            break;
        case OCT:
            selectTracesForOffsetContinuationTrajectoryAndDifferentialEvolution<<<dimGrid, threadCount>>>(
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::MDPNT]),
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::HLFOFFST]),
                CUDA_DEV_PTR(deviceParameterArray),
                traceCount,
                samplesPerTrace,
                individualsPerPopulation,
                dtInSeconds,
                apm,
                m0,
                h0,
                deviceUsedTraceMaskArray
            );
            break;
        default:
            throw invalid_argument("Invalid traveltime model");
    }

    CUDA_ASSERT(hipGetLastError());
    CUDA_ASSERT(hipDeviceSynchronize());

    CUDA_ASSERT(hipMemcpy(usedTraceMask.data(), deviceUsedTraceMaskArray, traceCount * sizeof(unsigned char), hipMemcpyDeviceToHost));
    CUDA_ASSERT(hipFree(deviceUsedTraceMaskArray));

    MEASURE_EXEC_TIME(copyTime, copyOnlySelectedTracesToDevice(usedTraceMask));

    LOGI("Execution time for copying traces is " << copyTime.count() << "s");
}

void CudaDifferentialEvolutionAlgorithm::setupRandomSeedArray() {

    Gather* gather = Gather::getInstance();

    deviceContext->activate();

    unsigned int samplesPerTrace = gather->getSamplesPerTrace();

    dim3 dimGrid(static_cast<int>(ceil(static_cast<float>(samplesPerTrace * individualsPerPopulation) / static_cast<float>(threadCount))));

    CUDA_ASSERT(hipMalloc(&st, samplesPerTrace * individualsPerPopulation * sizeof(hiprandState)));

    srand(static_cast<unsigned int>(time(NULL)));

    setupRandomSeed<<< dimGrid, threadCount >>>(st, rand(), individualsPerPopulation, samplesPerTrace);

    CUDA_ASSERT(hipGetLastError());

    CUDA_ASSERT(hipDeviceSynchronize());
}

void CudaDifferentialEvolutionAlgorithm::startAllPopulations() {

    Gather* gather = Gather::getInstance();

    unsigned int samplesPerTrace = gather->getSamplesPerTrace();
    unsigned int numberOfParameters = traveltime->getNumberOfParameters();

    dim3 dimGrid(static_cast<int>(ceil(static_cast<float>(samplesPerTrace * individualsPerPopulation) / static_cast<float>(threadCount))));

    startPopulations<<< dimGrid, threadCount >>>(
        CUDA_DEV_PTR(x),
        CUDA_DEV_PTR(min),
        CUDA_DEV_PTR(max),
        st,
        individualsPerPopulation,
        samplesPerTrace,
        numberOfParameters
    );

    CUDA_ASSERT(hipGetLastError());
    CUDA_ASSERT(hipDeviceSynchronize());

    fx->reset();
    fu->reset();
}

void CudaDifferentialEvolutionAlgorithm::mutateAllPopulations() {

    Gather* gather = Gather::getInstance();

    unsigned int samplesPerTrace = gather->getSamplesPerTrace();
    unsigned int numberOfParameters = traveltime->getNumberOfParameters();

    dim3 dimGrid(static_cast<int>(ceil(static_cast<float>(samplesPerTrace * individualsPerPopulation) / static_cast<float>(threadCount))));

    mutatePopulations<<< dimGrid, threadCount >>>(
        CUDA_DEV_PTR(v),
        CUDA_DEV_PTR(x),
        CUDA_DEV_PTR(min),
        CUDA_DEV_PTR(max),
        st,
        individualsPerPopulation,
        samplesPerTrace,
        numberOfParameters
    );

    CUDA_ASSERT(hipGetLastError());
    CUDA_ASSERT(hipDeviceSynchronize());
}

void CudaDifferentialEvolutionAlgorithm::crossoverPopulationIndividuals() {

    Gather* gather = Gather::getInstance();

    unsigned int samplesPerTrace = gather->getSamplesPerTrace();
    unsigned int numberOfParameters = traveltime->getNumberOfParameters();

    dim3 dimGrid(static_cast<int>(ceil(static_cast<float>(samplesPerTrace * individualsPerPopulation) / static_cast<float>(threadCount))));

    crossoverPopulations<<< dimGrid, threadCount >>>(
        CUDA_DEV_PTR(u),
        CUDA_DEV_PTR(x),
        CUDA_DEV_PTR(v),
        st,
        individualsPerPopulation,
        samplesPerTrace,
        numberOfParameters
    );

    CUDA_ASSERT(hipGetLastError());
    CUDA_ASSERT(hipDeviceSynchronize());
}

void CudaDifferentialEvolutionAlgorithm::advanceGeneration() {

    Gather* gather = Gather::getInstance();

    unsigned int samplesPerTrace = gather->getSamplesPerTrace();
    unsigned int numberOfParameters = traveltime->getNumberOfParameters();
    unsigned int numberOfCommonResults = traveltime->getNumberOfCommonResults();

    dim3 dimGrid(static_cast<int>(ceil(static_cast<float>(samplesPerTrace * individualsPerPopulation) / static_cast<float>(threadCount))));

    nextGeneration<<< dimGrid, threadCount >>>(
        CUDA_DEV_PTR(x),
        CUDA_DEV_PTR(fx),
        CUDA_DEV_PTR(u),
        CUDA_DEV_PTR(fu),
        individualsPerPopulation,
        samplesPerTrace,
        numberOfParameters,
        numberOfCommonResults
    );

    CUDA_ASSERT(hipGetLastError());
    CUDA_ASSERT(hipDeviceSynchronize());
}

void CudaDifferentialEvolutionAlgorithm::selectBestIndividuals(vector<float>& resultArrays) {

    Gather* gather = Gather::getInstance();

    unsigned int samplesPerTrace = gather->getSamplesPerTrace();
    unsigned int numberOfParameters = traveltime->getNumberOfParameters();
    unsigned int numberOfCommonResults = traveltime->getNumberOfCommonResults();

    dim3 dimGrid(static_cast<int>(ceil(static_cast<float>(samplesPerTrace) / static_cast<float>(threadCount))));

    switch (traveltime->getModel()) {
        case CMP:
            selectBestIndividualsForCommonMidPoint<<<dimGrid, threadCount>>>(
                CUDA_DEV_PTR(x),
                CUDA_DEV_PTR(fx),
                CUDA_DEV_PTR(deviceResultArray),
                individualsPerPopulation,
                samplesPerTrace,
                numberOfCommonResults
            );
            break;
        case ZOCRS:
            selectBestIndividualsForZeroOffsetCommonReflectionSurface<<<dimGrid, threadCount>>>(
                CUDA_DEV_PTR(x),
                CUDA_DEV_PTR(fx),
                CUDA_DEV_PTR(deviceResultArray),
                individualsPerPopulation,
                samplesPerTrace,
                numberOfCommonResults
            );
            break;
        case OCT:
            selectBestIndividualsForOffsetContinuationTrajectory<<<dimGrid, threadCount>>>(
                CUDA_DEV_PTR(x),
                CUDA_DEV_PTR(fx),
                CUDA_DEV_PTR(deviceResultArray),
                individualsPerPopulation,
                samplesPerTrace,
                numberOfCommonResults
            );
            break;
        default:
            throw invalid_argument("Invalid traveltime model");
    }

    CUDA_ASSERT(hipGetLastError());
    CUDA_ASSERT(hipDeviceSynchronize());

    deviceResultArray->pasteTo(resultArrays);
}
