#include "hip/hip_runtime.h"
#include "common/include/execution/Utils.hpp"
#include "common/include/output/Logger.hpp"
#include "cuda/include/execution/CudaUtils.hpp"
#include "cuda/include/semblance/data/CudaDataContainer.hpp"
#include "cuda/include/semblance/algorithm/CudaStretchFreeAlgorithm.hpp"
#include "cuda/include/semblance/kernel/base.h"
#include "cuda/include/semblance/kernel/stretch_free.h"
#include "cuda/include/semblance/kernel/common/stretch_free.cuh"
#include "cuda/include/semblance/kernel/cmp/common.cuh"
#include "cuda/include/semblance/kernel/cmp/stretch_free.cuh"
#include "cuda/include/semblance/kernel/zocrs/common.cuh"
#include "cuda/include/semblance/kernel/zocrs/stretch_free.cuh"
#include "cuda/include/semblance/kernel/oct/stretch_free.cuh"

#include <cmath>
#include <sstream>
#include <stdexcept>

using namespace std;

CudaStretchFreeAlgorithm::CudaStretchFreeAlgorithm(
    shared_ptr<Traveltime> traveltime,
    shared_ptr<DeviceContext> context,
    DataContainerBuilder* dataBuilder,
    const vector<string>& files
) : StretchFreeAlgorithm(traveltime, context, dataBuilder, files) {
}

void CudaStretchFreeAlgorithm::computeSemblanceAtGpuForMidpoint(float m0) {

    LOGI("Computing semblance for m0 = " << m0);

    if (!filteredTracesCount) {
        LOGI("No trace has been selected for m0 = " << m0 << ". Skipping.");
        return;
    }

    unsigned int totalNumberOfParameters = getTotalNumberOfParameters();
    unsigned int numberOfCommonResults = traveltime->getNumberOfCommonResults();

    Gather* gather = Gather::getInstance();

    unsigned int samplesPerTrace = gather->getSamplesPerTrace();
    float dtInSeconds = gather->getSamplePeriodInSeconds();
    int tauIndexDisplacement = gather->getTauIndexDisplacement();
    unsigned int windowSize = gather->getWindowSize();

    dim3 dimGrid(static_cast<int>(ceil(static_cast<float>(totalNumberOfParameters * samplesPerTrace) / static_cast<float>(threadCount))));

    switch (traveltime->getModel()) {
        case CMP: {
            computeSemblancesForCommonMidPoint<<< dimGrid, threadCount >>>(
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::FILT_SAMPL]),
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::FILT_HLFOFFST_SQ]),
                filteredTracesCount,
                samplesPerTrace,
                dtInSeconds,
                tauIndexDisplacement,
                windowSize,
                CUDA_DEV_PTR(nonStretchFreeParameters[m0]),
                CUDA_DEV_PTR(deviceParameterArray),
                totalNumberOfParameters,
                CUDA_DEV_PTR(commonResultDeviceArrayMap[SemblanceCommonResult::SEMBL]),
                CUDA_DEV_PTR(commonResultDeviceArrayMap[SemblanceCommonResult::STACK])
            );
            break;
        }
        case ZOCRS: {
            computeSemblancesForZeroOffsetCommonReflectionSurface<<< dimGrid, threadCount >>>(
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::FILT_SAMPL]),
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::FILT_MDPNT]),
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::FILT_HLFOFFST_SQ]),
                filteredTracesCount,
                samplesPerTrace,
                m0,
                dtInSeconds,
                tauIndexDisplacement,
                windowSize,
                CUDA_DEV_PTR(nonStretchFreeParameters[m0]),
                CUDA_DEV_PTR(deviceParameterArray),
                totalNumberOfParameters,
                CUDA_DEV_PTR(commonResultDeviceArrayMap[SemblanceCommonResult::SEMBL]),
                CUDA_DEV_PTR(commonResultDeviceArrayMap[SemblanceCommonResult::STACK])
            );
            break;
        }
        case OCT: {
            break;
        }
        default:
            throw invalid_argument("Invalid traveltime model");
    }

    dim3 dimGridBest(static_cast<int>(ceil(static_cast<float>(samplesPerTrace) / static_cast<float>(threadCount))));

    selectBestSemblances<<< dimGridBest, threadCount >>>(
        CUDA_DEV_PTR(commonResultDeviceArrayMap[SemblanceCommonResult::SEMBL]),
        CUDA_DEV_PTR(commonResultDeviceArrayMap[SemblanceCommonResult::STACK]),
        CUDA_DEV_PTR(deviceParameterArray),
        totalNumberOfParameters,
        samplesPerTrace,
        CUDA_DEV_PTR(deviceResultArray)
    );

    CUDA_ASSERT(hipDeviceSynchronize());
    CUDA_ASSERT(hipGetLastError());
}

void CudaStretchFreeAlgorithm::selectTracesToBeUsedForMidpoint(float m0) {

    Gather* gather = Gather::getInstance();

    unsigned int traceCount = gather->getTotalTracesCount();
    unsigned int totalNumberOfParameters = getTotalNumberOfParameters();

    unsigned int samplesPerTrace = gather->getSamplesPerTrace();
    float dtInSeconds = gather->getSamplePeriodInSeconds();
    int tauIndexDisplacement = gather->getTauIndexDisplacement();
    unsigned int windowSize = gather->getWindowSize();
    float apm = gather->getApm();

    vector<unsigned char> usedTraceMask(traceCount);

    unsigned char* deviceUsedTraceMaskArray;
    CUDA_ASSERT(hipMalloc((void **) &deviceUsedTraceMaskArray, traceCount * sizeof(unsigned char)));
    CUDA_ASSERT(hipMemset(deviceUsedTraceMaskArray, 0, traceCount * sizeof(unsigned char)))

    dim3 dimGrid(static_cast<int>(ceil(static_cast<float>(traceCount) / static_cast<float>(threadCount))));

    LOGI("Using " << dimGrid.x << " blocks for traces filtering (threadCount = "<< threadCount << ")");

    chrono::duration<double> copyTime = chrono::duration<double>::zero();

    switch (traveltime->getModel()) {
        case CMP:
            selectTracesForCommonMidPoint<<<dimGrid, threadCount>>>(
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::MDPNT]),
                traceCount,
                deviceUsedTraceMaskArray,
                m0
            );
            break;
        case ZOCRS:
            selectTracesForZeroOffsetCommonReflectionSurface<<<dimGrid, threadCount>>>(
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::MDPNT]),
                traceCount,
                deviceUsedTraceMaskArray,
                m0,
                apm
            );
            break;
        case OCT:
            selectTracesForOffsetContinuationTrajectory<<<dimGrid, threadCount>>>(
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::MDPNT]),
                CUDA_DEV_PTR(deviceFilteredTracesDataMap[GatherData::HLFOFFST]),
                CUDA_DEV_PTR(nonStretchFreeParameters[m0]),
                traceCount,
                samplesPerTrace,
                dtInSeconds,
                apm,
                m0,
                traveltime->getReferenceHalfoffset(),
                deviceUsedTraceMaskArray
            );
            break;
        default:
            throw invalid_argument("Invalid traveltime model");
    }

    CUDA_ASSERT(hipDeviceSynchronize());

    CUDA_ASSERT(hipGetLastError());

    CUDA_ASSERT(hipMemcpy(usedTraceMask.data(), deviceUsedTraceMaskArray, traceCount * sizeof(unsigned char), hipMemcpyDeviceToHost));

    CUDA_ASSERT(hipFree(deviceUsedTraceMaskArray));

    MEASURE_EXEC_TIME(copyTime, copyOnlySelectedTracesToDevice(usedTraceMask));

    LOGI("Execution time for copying traces is " << copyTime.count() << "s");
}
